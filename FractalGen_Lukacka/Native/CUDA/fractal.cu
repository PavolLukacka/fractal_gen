#include "hip/hip_runtime.h"
﻿#include <cstdint>
#include <hip/hip_runtime.h>

struct Params {
    double xCenter;
    double yCenter;
    double zoom;
    int    width;
    int    height;
    int    maxIter;
    int    palette;       // 0=gray,1=rainbow,2=fire
};

__device__ int mandelbrot(double cx, double cy, int maxIter)
{
    double x = 0.0, y = 0.0, xx = 0.0, yy = 0.0;
    int i = 0;
    while (xx + yy <= 4.0 && i < maxIter)
    {
        y = 2 * x * y + cy;
        x = xx - yy + cx;
        xx = x * x;
        yy = y * y;
        ++i;
    }
    return i;
}

__global__ void renderKernel(const Params p, uint8_t* rgb)
{
    int px = blockIdx.x * blockDim.x + threadIdx.x;
    int py = blockIdx.y * blockDim.y + threadIdx.y;
    if (px >= p.width || py >= p.height) return;

    double scale = 1.5 / p.zoom;   // view fits in [-1.5,1.5] at zoom 1
    double xMin = p.xCenter - scale;
    double yMin = p.yCenter - scale;

    double cx = xMin + (double)px / p.width * 2.0 * scale;
    double cy = yMin + (double)py / p.height * 2.0 * scale;

    int iter = mandelbrot(cx, cy, p.maxIter);

    // --- very small, fixed palettes -------------
    uint8_t r, g, b;
    if (iter == p.maxIter) { r = g = b = 0; } // in the set → black
    else
    {
        double t = (double)iter / p.maxIter;   // 0…1
        switch (p.palette)
        {
        case 0: r = g = b = (uint8_t)(t * 255); break;           // grayscale
        case 1: r = (uint8_t)(t * 255); g = (uint8_t)((1 - t) * 255); b = 128; break; // rainbow-ish
        default: r = (uint8_t)(t * 255); g = (uint8_t)(t * t * 255); b = 0;  // fire
        }
    }

    int idx = (py * p.width + px) * 3;
    rgb[idx] = r;
    rgb[idx + 1] = g;
    rgb[idx + 2] = b;
}

// ----------- exported C function ---------------
extern "C" __declspec(dllexport)
int GenerateFractal(const Params * pHost, uint8_t * rgbHost)
{
    Params  p = *pHost;                  // copy to device later
    size_t  imgBytes = p.width * p.height * 3;

    uint8_t* dRgb = nullptr;
    Params* dPar = nullptr;

    hipError_t err = hipSuccess;
    if ((err = hipMalloc(&dRgb, imgBytes)) != hipSuccess) return err;
    if ((err = hipMalloc(&dPar, sizeof(Params))) != hipSuccess) return err;

    hipMemcpy(dPar, &p, sizeof(Params), hipMemcpyHostToDevice);

    dim3 block(16, 16);
    dim3 grid((p.width + 15) / 16, (p.height + 15) / 16);
    renderKernel << <grid, block >> > (*dPar, dRgb);
    if ((err = hipGetLastError()) != hipSuccess) return err;

    hipMemcpy(rgbHost, dRgb, imgBytes, hipMemcpyDeviceToHost);

    hipFree(dRgb); hipFree(dPar);
    return 0;           // success
}
